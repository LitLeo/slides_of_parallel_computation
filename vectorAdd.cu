
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

using namespace std;

__global__ void vectorAddKer(int *d_A, int *d_B, int *d_C, int size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if(index >= size) {
        return;
    }
    d_C[index] = d_A[index] + d_B[index];
}



int main(int argc, char const *argv[]) {
    int size = 0;
    if(argc != 2) {
        size = 1024;
    } else {
        size = atoi(argv[1]);
    }

    int *h_A, *h_B, *h_C;
    int *d_A, *d_B, *d_C;

    h_A = new int[size];
    h_B = new int[size];
    h_C = new int[size];

    for(unsigned i = 0; i < size; ++i) {
        h_A[i] = i;
        h_B[i] = size - i;
    }

    hipEvent_t start, stop;
    float elapsedTime = 0.0;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipMalloc(&d_A, size * sizeof(int));
    hipMalloc(&d_B, size * sizeof(int));
    hipMalloc(&d_C, size * sizeof(int));

    hipMemcpy(d_A, h_A, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size * sizeof(int), hipMemcpyHostToDevice);

    dim3 gridsize, blocksize;
    blocksize.x = 256;
    gridsize.x = (size + blocksize.x - 1) / blocksize.x;

    vectorAddKer<<<gridsize, blocksize>>>(d_A, d_B, d_C, size);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);

    cout << "time=" << elapsedTime << endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(h_C, d_C, size * sizeof(int), hipMemcpyDeviceToHost);

    for(unsigned i = 0; i < size; ++i) {
        printf("%d + %d = %d\n", h_A[i], h_B[i], h_C[i]);
    }

    return 0;
}